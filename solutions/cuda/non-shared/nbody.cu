#include "hip/hip_runtime.h"
#include "global.h"

double drandom(double x,double y)
{
  double out = VRG(gen);
  out = x + (y - x)*out;
  return out;
}

void write_state(int timestep,const double* x)
{
  int i,zero = 0;
  std::stringstream sstream;
  sstream << "nbody_" << timestep << ".mol";
  std::string filename = sstream.str();
  std::ofstream s(filename.c_str());
  s << "nbody_" << timestep << std::endl;
  s << "  MOE2000" << std::endl;
  s << std::endl;
  s << std::setw(3) << NP << std::setw(3) << zero << " 0  0  0  0  0  0  0  0   1 V2000" << std::endl;
  for(i=0; i<NP; ++i) {
    s << std::setw(10) << std::setprecision(4) << std::setiosflags(std::ios::fixed) << x[3*i] << std::setw(10) << std::setprecision(4) << std::setiosflags(std::ios::fixed) << x[3*i+1] << std::setw(10) << std::setprecision(4) << std::setiosflags(std::ios::fixed) << x[3*i+2] << " " << std::setw(3) << std::setiosflags(std::ios::left) << "C" << std::resetiosflags(std::ios::left) << " 0  0  0  0  0  0  0  0  0  0  0  0" << std::endl;
  }
  s << "M  END" << std::endl;
  s << "$$$$" << std::endl;
  s.close();
}

void boundary_conditions(double* x)
{
  int i,j;
  double xv,dsize;

  if (!finite_domain) return;

  for(i=0; i<NP; ++i) {
    for(j=0; j<3; ++j) {
      xv = x[3*i+j];
      if (xv < L[2*j]) {
        dsize = L[2*j+1] - L[2*j];
        do {
          xv += dsize;
          if (xv > L[2*j]) break;
        } while(true);
        x[3*i+j] = xv;
      }
      else if (xv > L[2*j+1]) {
        dsize = L[2*j+1] - L[2*j];
        do {
          xv -= dsize;
          if (xv < L[2*j+1]) break;
        } while(true);
        x[3*i+j] = xv;
      }
    }
  }
}

__global__ void boundary_conditions_gpu(int NP, bool finite_domain, const double *L, double *x)
{
   int i,j;
   double xv,dsize;
   i=blockIdx.x*blockDim.x + threadIdx.x;

   if(finite_domain){
   for(j=0; j<3; ++j) {
   	xv = x[3*i+j];
      	if (xv < L[2*j]) { 
      		dsize = L[2*j+1] - L[2*j];
        	do { 
          		xv += dsize;
          		if (xv > L[2*j]) break;
        	} while(true);
        	x[3*i+j] = xv;
      	}
	else if (xv > L[2*j+1]) {
        	dsize = L[2*j+1] - L[2*j];
        	do { 
          		xv -= dsize;
          		if (xv < L[2*j+1]) break;
        	} while(true);
        	x[3*i+j] = xv;
      	}

   }

   __syncthreads();
  }
}

void compute_acceleration(const double* x,const double* mass,double* acc)
{
  int i,j,k;
  double delta,rij,pfactor,sum[3];

  for(i=0; i<NP; ++i) {
    for(j=0; j<3; ++j) {
      sum[j] = 0.0;
    }
    for(j=0; j<NP; ++j) {
      if (i == j) continue;
      delta = 0.0;
      for(k=0; k<3; ++k) {
        delta += (x[3*i+k] - x[3*j+k])*(x[3*i+k] - x[3*j+k]);
      }
      rij = std::sqrt(epsilon + delta);
      pfactor = mass[j]/(rij*rij*rij);
      for(k=0; k<3; ++k) {
        sum[k] += pfactor*(x[3*i+k] - x[3*j+k]);
      }
    }
    for(j=0; j<3; ++j) {
      acc[3*i+j] = -sum[j];
    }
  }
}

__global__ void compute_acceleration_gpu(int NP,double epsilon,const double *x,const double *mass,double *acc)
{
   int i,j,k;
   double delta,rij,pfactor,sum[3];
   i=blockIdx.x*blockDim.x + threadIdx.x;
   for(k=0; k<3; ++k) sum[k] = 0.0;
   for(j=0;j<NP;++j){
	if(i!=j){
		delta = 0.0;
		for(k=0; k<3; ++k) delta += (x[3*i+k] - x[3*j+k])*(x[3*i+k] - x[3*j+k]);
	//	rij = std::sqrt(epsilon + delta);
		rij = __dsqrt_rd (epsilon + delta);
		pfactor = mass[j]/(rij*rij*rij);
		for(k=0; k<3; ++k) sum[k] += pfactor*(x[3*i+k] - x[3*j+k]);	
	}
   }
   __syncthreads();
   for(k=0; k<3; ++k) acc[3*i+k] = -sum[k];
   __syncthreads();
}

double compute_energy(const double* x,const double* v,const double* mass)
{
  int i,j,k;
  double delta,T = 0.0,U = 0.0;

  // First the kinetic energy...
  for(i=0; i<NP; ++i) {
    delta = 0.0;
    for(j=0; j<3; ++j) {
      delta += v[3*i+j]*v[3*i+j];
    }
    T += mass[i]*delta;
  }
 
  // Now the potential energy
  for(i=0; i<NP; ++i) {
    for(j=1+i; j<NP; ++j) {
      delta = 0.0;
      for(k=0; k<3; ++k) {
        delta += (x[3*i+k] - x[3*j+k])*(x[3*i+k] - x[3*j+k]);
      }
      U += mass[i]*mass[j]/std::sqrt(epsilon + delta); 
    }
  }
  return (0.5*T - U);  
}


__global__ void compute_energy_gpu(int NP,const double epsilon,const double *x,const double *v,const double *mass,double *E)
{
  int i,j,k;
  double delta;
  i=blockIdx.x*blockDim.x + threadIdx.x;

  // First the kinetic energy...
  delta = 0.0;
  for(j=0; j<3; ++j) delta += v[3*i+j]*v[3*i+j];
  E[i] = 0.5*mass[i]*delta;

  // Now the potential energy
  for(j=1+i; j<NP; ++j) {
  	delta = 0.0;
	for(k=0; k<3; ++k) delta += (x[3*i+k] - x[3*j+k])*(x[3*i+k] - x[3*j+k]);
	E[i] -= mass[i]*mass[j]/std::sqrt(epsilon + delta);
  }
  __syncthreads(); 
}

double compute_kinetic_energy(const double* x,const double* v,const double* mass)
{
  int i,j;
  double delta,T = 0.0;

  for(i=0; i<NP; ++i) {
    delta = 0.0;
    for(j=0; j<3; ++j) {
      delta += v[3*i+j]*v[3*i+j];
    }
    T += mass[i]*delta;
  }
 
  return 0.5*T;  
}

double compute_potential_energy(const double* x,const double* v,const double* mass)
{
  int i,j,k;
  double delta,U = 0.0;

  for(i=0; i<NP; ++i) {
    for(j=1+i; j<NP; ++j) {
      delta = 0.0;
      for(k=0; k<3; ++k) {
        delta += (x[3*i+k] - x[3*j+k])*(x[3*i+k] - x[3*j+k]);
      }
      U += mass[i]*mass[j]/std::sqrt(epsilon + delta); 
    }
  }
  return U;  
}

void compute_center_of_mass(const double* x, const double* mass, double* center)
{
  int i,j;
  double total_mass = 0.0;

  for(j=0; j<3; ++j) {
    center[j] = 0.0;
  }
  for(i=0; i<NP; ++i) {
    for(j=0; j<3; ++j) {
      center[j] += x[3*i+j]*mass[i];
    }
    total_mass += mass[i];
  }
  for(j=0; j<3; ++j) {
    center[j] /= total_mass;
  }
}

void center_particles(double* x, const double* mass)
{
  int i,j;
  double center[3];
  compute_center_of_mass(x,mass,center);
  for(i=0; i<NP; ++i) {
    for(j=0; j<3; ++j) {
      x[3*i+j] -= center[j];
    }
  }
}

__global__ void get_new_coordinates_gpu(const double dt,const double *x,const double *v, const double *acc, double *xnew)
{
  int i=blockIdx.x*blockDim.x + threadIdx.x;
  for(int j=0; j<3; ++j) {
  	xnew[3*i+j] = x[3*i+j] + dt*v[3*i+j] + 0.5*dt*dt*acc[3*i+j];
//	xnew[3*i+j] = x[3*i+j] dt*v[3*i+j];
  }
  __syncthreads();
  
}

__global__ void get_new_velocities_gpu(const double dt,const double *v, const double *temp,const double *acc, double *vnew)
{
  int i=blockIdx.x*blockDim.x + threadIdx.x; 
      for(int j=0; j<3; ++j) {
        vnew[3*i+j] = v[3*i+j] + 0.5*dt*(acc[3*i+j] + temp[3*i+j]);
      }
  __syncthreads();
}

__global__ void update_variables_gpu(const double *xnew, const double *vnew, const double *temp, double *x, double *v, double *acc)
{
  int i=blockIdx.x*blockDim.x + threadIdx.x;
      for(int j=0; j<3; ++j) {
        x[3*i+j] = xnew[3*i+j];
        v[3*i+j] = vnew[3*i+j];
        acc[3*i+j] = temp[3*i+j];
      }
  __syncthreads();  
}

__global__ void kernel_RK1_gpu(const int NP,const double dt,const double *x,const double *v,const double *acc,double *k1,double *temp)
{
  int i=blockIdx.x*blockDim.x + threadIdx.x;
      for(int j=0; j<3; ++j) {
        k1[3*i+j] = v[3*i+j];
        k1[3*NP+3*i+j] = acc[3*i+j];
        temp[3*i+j] = x[3*i+j] + 0.5*dt*k1[3*i+j];
      }
  __syncthreads();
}

__global__ void kernel_RK2_gpu(const int NP,const double dt,const double *x,const double *v,const double *acc,double *k2,double *temp)
{ 
  int i=blockIdx.x*blockDim.x + threadIdx.x;
     for(int j=0; j<3; ++j) {
        k2[3*i+j] = (1.0 + 0.5*dt)*v[3*i+j];
        k2[3*NP+3*i+j] = acc[3*i+j];
        temp[3*i+j] = x[3*i+j] + 0.5*dt*k2[3*i+j];
      }
  __syncthreads();
}

__global__ void kernel_RK3_gpu(const int NP,const double dt,const double *x,const double *v,const double *acc,double *k3,double *temp)
{
  int i=blockIdx.x*blockDim.x + threadIdx.x;
  for(int j=0; j<3; ++j) {
        k3[3*i+j] = (1.0 + 0.5*dt + 0.25*dt*dt)*v[3*i+j];
        k3[3*NP+3*i+j] = acc[3*i+j];
        temp[3*i+j] = x[3*i+j] + dt*k3[3*i+j];
      }
  __syncthreads();
}

__global__ void kernel_RK4_gpu(const int NP,const double dt,const double *x,const double *v,const double *acc,double *k4)
{
  int i=blockIdx.x*blockDim.x + threadIdx.x;
  for(int j=0; j<3; ++j) {
        k4[3*i+j] = (1.0 + dt + 0.5*dt*dt + 0.25*dt*dt*dt)*v[3*i+j];
        k4[3*NP+3*i+j] = acc[3*i+j];
      }
  __syncthreads();
}

__global__ void get_newXV_RK_gpu(const int NP,const double dt,const double *x,const double *v,const double *k1,const double *k2,const double *k3,const double *k4, double *xnew, double *vnew)
{
  int i=blockIdx.x*blockDim.x + threadIdx.x;
      for(int j=0; j<3; ++j) {
        xnew[3*i+j] = x[3*i+j] + dt*(k1[3*i+j] + 2.0*k2[3*i+j] + 2.0*k3[3*i+j]+k4[3*i+j])/6.0;
        vnew[3*i+j] = v[3*i+j] + dt*(k1[3*NP+3*i+j] + 2.0*k2[3*NP+3*i+j] + 2.0*k3[3*NP+3*i+j] + k4[3*NP+3*i+j])/6.0;
      }
  __syncthreads();
}

__global__ void update_variables_RK_gpu(const double *xnew, const double *vnew, double *x,double *v)
{
  int i=blockIdx.x*blockDim.x + threadIdx.x;
  for(int j=0; j<3; ++j) {
        x[3*i+j] = xnew[3*i+j];
        v[3*i+j] = vnew[3*i+j];
      }
  __syncthreads();
}


void integrate()
{
  int i,j,l;
  double x[3*NP],xnew[3*NP],v[3*NP];
//  double vnew[3*NP],acc[3*NP],temp[3*NP];
  double mass[NP];
  double K,U,alpha;

  // Define GPU variables
  double *d_x, *d_xnew, *d_v, *d_vnew;
  double *d_mass, *d_acc, *d_temp;
  double *E, *d_E;
  double *d_L;
  double Ef;

  // Allocate GPU variables
  size_t memSize=sizeof(double)*NP*3;
  hipMalloc((void**)&d_x,sizeof(double)*NP*3);
  hipMalloc((void**)&d_xnew,sizeof(double)*NP*3);
  hipMalloc((void**)&d_v,sizeof(double)*NP*3);
  hipMalloc((void**)&d_vnew,sizeof(double)*NP*3);
  hipMalloc((void**)&d_acc,sizeof(double)*NP*3);
  hipMalloc((void**)&d_temp,sizeof(double)*NP*3);
  hipMalloc((void**)&d_mass,sizeof(double)*NP);
  hipMalloc((void**)&d_E,sizeof(double)*NP);
  hipMalloc((void**)&d_L,6*sizeof(double));

  E = (double*)malloc(sizeof(double)*NP);

  // Assign initial values...
  for(i=0; i<NP; ++i) {
    for(j=0; j<3; ++j) {
      // Initial position and speed
      x[3*i+j] = drandom(L[2*j],L[2*j+1]);
      v[3*i+j] = drandom(-0.2,0.2);
    }
  }
  // Assign random mass
  for(i=0; i<NP; ++i) {
    mass[i] = drandom(low_mass,high_mass);
  }

  // Add a rotation around the z axis
  for(i=0; i<NP; ++i) {
    v[3*i+1] += x[3*i+0]/10.0;
    v[3*i+0] -= x[3*i+1]/10.0;
  }

  if (center_masses) {
    // Set the center of mass and it's speed to 0
    center_particles(x, mass);
    center_particles(v, mass);
  }

  if (bounded_state) {
    // Make sure that the total energy of the system is negative so particle don't fly in the distance
    // Set the kinetic energy to half the potential energy
    U = compute_potential_energy(x,v,mass);
    K = compute_kinetic_energy(x,v,mass);
    alpha = std::sqrt(U/(2.0*K));

    for(i=0; i<NP; ++i) {
      for(j=0; j<3; ++j) {
        v[3*i+j] *= alpha;
      }
    }
  }

  write_state(0,x);
  std::cout << "0.0  " << compute_energy(x,v,mass)/double(NP) << std::endl;

#ifdef VERLET
  clock_t start, end;
     double cpu_time_used;
     start = clock();
  int numThreadsPerBlock=256;
  int numBlocks=NP/numThreadsPerBlock;
  dim3 dimGrid(numBlocks);
  dim3 dimBlock(numThreadsPerBlock);

  // Copy data to GPU
  hipMemcpy(d_x,&x[0],memSize,hipMemcpyHostToDevice);
  hipMemcpy(d_mass,&mass[0],sizeof(double)*NP,hipMemcpyHostToDevice);
  hipMemcpy(d_v,&v[0],memSize,hipMemcpyHostToDevice);
  hipMemcpy(d_L,&L[0],6*sizeof(double),hipMemcpyHostToDevice);

  // Compute acceleration on GPU 
  compute_acceleration_gpu <<<dimGrid, dimBlock>>>(NP,epsilon,d_x,d_mass,d_acc);


  for(l=1; l<=NT; ++l) {

  // Get new coordinates done on GPU
  get_new_coordinates_gpu <<<dimGrid, dimBlock>>>(dt,d_x,d_v,d_acc,d_xnew);


    // Apply boundary conditions 
    boundary_conditions_gpu <<<dimGrid, dimBlock>>> (NP,finite_domain,d_L,d_xnew); 

    // Compute acceleration on GPU 
    compute_acceleration_gpu <<<dimGrid, dimBlock>>>(NP,epsilon,d_xnew,d_mass,d_temp);


    // Get new velocities done on GPU
    get_new_velocities_gpu <<<dimGrid, dimBlock>>> (dt,d_v,d_temp,d_acc,d_vnew);

    // Print out the system's total energy per particle (should be fairly constant)
    if (l%100 == 0) {
      compute_energy_gpu <<<dimGrid, dimBlock>>> (NP,epsilon,d_x,d_v,d_mass,d_E);
      hipMemcpy(E,d_E,sizeof(double)*NP,hipMemcpyDeviceToHost);
      Ef=0.0;
      for(int i=0;i<NP;i++) Ef+=E[i];
      std::cout << dt*double(l) << "  " << Ef/double(NP) << std::endl;

    }
    if ((l % write_freq) == 0){
	hipMemcpy(&xnew[0],d_xnew,memSize,hipMemcpyDeviceToHost);
	write_state(l,xnew);
    }

    // Now update the arrays on GPU
    update_variables_gpu <<<dimGrid, dimBlock>>> (d_xnew,d_vnew,d_temp,d_x,d_v,d_acc);

  }
  end = clock();
  cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
  printf("Exec time = %f\n",cpu_time_used);
#else
  // Fourth-order Runge-Kutta
//  double k1[6*NP],k2[6*NP],k3[6*NP],k4[6*NP];

  clock_t start, end;
  double cpu_time_used;
  start = clock();
  int numThreadsPerBlock=256;
  int numBlocks=NP/numThreadsPerBlock;
  dim3 dimGrid(numBlocks);
  dim3 dimBlock(numThreadsPerBlock);

  double *d_k1, *d_k2, *d_k3, *d_k4;
  hipMalloc((void**)&d_k1,sizeof(double)*6*NP);
  hipMalloc((void**)&d_k2,sizeof(double)*6*NP);
  hipMalloc((void**)&d_k3,sizeof(double)*6*NP);
  hipMalloc((void**)&d_k4,sizeof(double)*6*NP);

  // Copy data to GPU
  hipMemcpy(d_x,&x[0],memSize,hipMemcpyHostToDevice);
  hipMemcpy(d_mass,&mass[0],sizeof(double)*NP,hipMemcpyHostToDevice);
  hipMemcpy(d_v,&v[0],memSize,hipMemcpyHostToDevice);
  hipMemcpy(d_L,&L[0],6*sizeof(double),hipMemcpyHostToDevice);

  
  for(l=1; l<=NT; ++l) {
    // Compute acceleration on GPU 
    compute_acceleration_gpu <<<dimGrid, dimBlock>>>(NP,epsilon,d_x,d_mass,d_acc);
//    hipMemcpy(&acc[0],d_acc,memSize,hipMemcpyDeviceToHost);

    // Compute coefficients K1 on GPU
    kernel_RK1_gpu <<< dimGrid, dimBlock >>> (NP,dt,d_x,d_v,d_acc,d_k1,d_temp);
//    hipMemcpy(&temp[0],d_temp,memSize,hipMemcpyDeviceToHost);
//    hipMemcpy(&k1[0],d_k1,sizeof(double)*6*NP,hipMemcpyDeviceToHost);

    // Compute acceleration on GPU 
    compute_acceleration_gpu <<<dimGrid, dimBlock>>>(NP,epsilon,d_temp,d_mass,d_acc);
//    hipMemcpy(&acc[0],d_acc,memSize,hipMemcpyDeviceToHost);

    // Compute coefficients K2 on GPU
    kernel_RK2_gpu <<< dimGrid, dimBlock >>> (NP,dt,d_x,d_v,d_acc,d_k2,d_temp);
//    hipMemcpy(&temp[0],d_temp,memSize,hipMemcpyDeviceToHost);
//    hipMemcpy(&k2[0],d_k2,sizeof(double)*6*NP,hipMemcpyDeviceToHost);

    // Compute acceleration on GPU 
    compute_acceleration_gpu <<<dimGrid, dimBlock>>>(NP,epsilon,d_temp,d_mass,d_acc);
//    hipMemcpy(&acc[0],d_acc,memSize,hipMemcpyDeviceToHost);

    // Compute coefficients K3 on GPU
    kernel_RK3_gpu <<< dimGrid, dimBlock >>> (NP,dt,d_x,d_v,d_acc,d_k3,d_temp);
//    hipMemcpy(&temp[0],d_temp,memSize,hipMemcpyDeviceToHost);
//    hipMemcpy(&k3[0],d_k3,sizeof(double)*6*NP,hipMemcpyDeviceToHost);

    // Compute acceleration on GPU 
    compute_acceleration_gpu <<<dimGrid, dimBlock>>>(NP,epsilon,d_temp,d_mass,d_acc);
//    hipMemcpy(&acc[0],d_acc,memSize,hipMemcpyDeviceToHost);
    
    // Compute coefficients K4 on GPU
    kernel_RK4_gpu <<< dimGrid, dimBlock >>> (NP,dt,d_x,d_v,d_acc,d_k4);
//    hipMemcpy(&k4[0],d_k4,sizeof(double)*6*NP,hipMemcpyDeviceToHost);


    // Get new xnew and vnew on GPU
    get_newXV_RK_gpu <<< dimGrid, dimBlock >>> (NP,dt,d_x,d_v,d_k1,d_k2,d_k3,d_k4, d_xnew, d_vnew);
//    hipMemcpy(&xnew[0],d_xnew,memSize,hipMemcpyDeviceToHost);
//    hipMemcpy(&vnew[0],d_vnew,memSize,hipMemcpyDeviceToHost);


    // Apply boundary conditions 
    boundary_conditions_gpu <<<dimGrid, dimBlock>>> (NP,finite_domain,d_L,d_xnew);
//    hipMemcpy(&xnew[0],d_xnew,memSize,hipMemcpyDeviceToHost);


    // Print out the system's total energy per particle (should be fairly constant)
    if (l%100 == 0) {
      compute_energy_gpu <<<dimGrid, dimBlock>>> (NP,epsilon,d_x,d_v,d_mass,d_E);
      hipMemcpy(E,d_E,sizeof(double)*NP,hipMemcpyDeviceToHost);
      Ef=0.0;
      for(int i=0;i<NP;i++) Ef+=E[i];
      std::cout << dt*double(l) << "  " << Ef/double(NP) << std::endl;
    //  std::cout << dt*double(l) << "  " << compute_energy(xnew,vnew,mass)/double(NP) << std::endl;
    }
    if ((l % write_freq) == 0){
	hipMemcpy(&xnew[0],d_xnew,memSize,hipMemcpyDeviceToHost);
	write_state(l,xnew);
    }

    // Now update the arrays on GPU
    update_variables_RK_gpu <<<dimGrid, dimBlock>>> (d_xnew,d_vnew,d_x,d_v);

  }
  end = clock();
  cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
  printf("Exec time = %f\n",cpu_time_used);
#endif
  write_state(NT,x);
  hipFree(d_x);
  hipFree(d_v);
  hipFree(d_acc);
  hipFree(d_xnew);
  hipFree(d_vnew);
  hipFree(d_temp);
  hipFree(d_mass);
  hipFree(d_E);
  hipFree(d_L);
}

void read_parameters(const char* filename)
{
  unsigned int i,bpoint;
  double tvalue;
  std::string line,name,value;

  std::ifstream s(filename);
  if (!s.is_open()) {
    // If the file doesn't exist, we need to exit...
    std::cout << "The file " << filename << " cannot be found!" << std::endl;
    std::exit(1);
  }

  // Loop through all lines in the parameter file
  while(std::getline(s,line)) {
    // If it's an empty line, continue
    if (line.empty()) continue;
    // If the line begins with a #, ignore it
    if (line[0] == '#') continue;
    // If there's no equals sign in this line, continue
    if (line.find('=') == std::string::npos) continue;
    // Assumes that the equals sign can only occur once in 
    // the line
    bpoint = 0;
    name = "";
    for(i=0; i<line.size(); ++i) {
      if (line[i] == ' ') continue;
      if (line[i] == '=') {
        bpoint = i;
        break;
      }
      name += line[i];
    }
    value = "";
    for(i=1+bpoint; i<line.size(); ++i) {
      if (line[i] == ' ') continue;
      value += line[i];
    }
    // Now that we have the parameter name, see if it matches
    // any of the known parameters. If so, read in the value and
    // assign it
    if (name == "nparticle") {
      NP = stoi(value);
    }
    else if (name == "max_time") {
      tvalue = stod(value);
    }
    else if (name == "seed") {
      seed = stoi(value);
    }
    else if (name == "timestep") {
      dt = stod(value);
    }
    else if (name == "epsilon") {
      epsilon = stod(value);
    }
    else if (name == "min_mass") {
      low_mass = stod(value);
    }
    else if (name == "max_mass") {
      high_mass = stod(value);
    }
    else if (name == "write_frequency") {
      write_freq = stoi(value);
    }
    else if (name == "finite_domain") {
      finite_domain = (value == "yes") ? true : false;
    }
    else if (name == "center_of_mass") {
      center_masses = (value == "yes") ? true : false;
    }
    else if (name == "bound_state") {
      bounded_state = (value == "yes") ? true : false;
    }
    else if (name == "xmin") {
      L[0] = stod(value);
    }
    else if (name == "xmax") {
      L[1] = stod(value);
    }
    else if (name == "ymin") {
      L[2] = stod(value);
    }
    else if (name == "ymax") {
      L[3] = stod(value);
    }
    else if (name == "zmin") {
      L[4] = stod(value);
    }
    else if (name == "zmax") {
      L[5] = stod(value);
    }
  }
  s.close();
  // Sanity checks
  assert(tvalue > std::numeric_limits<double>::epsilon());
  assert(NP > 1);
  assert(dt > std::numeric_limits<double>::epsilon());
  assert(epsilon > std::numeric_limits<double>::epsilon() && epsilon < 0.1);
  assert(write_freq > 0);
  assert(low_mass > std::numeric_limits<double>::epsilon());
  assert(high_mass >= low_mass);
  assert(seed >= 0);
  for(int i=0; i<3; ++i) {
    assert(L[2*i+1] > L[2*i]);
  }
  if (seed == 0) seed = std::time(NULL);
  gen.seed(seed);
  NT = int(tvalue/dt);
}

int main(int argc,char** argv)
{
  if (argc > 2) {
    std::cerr << "Usage: ./nbody parameters.txt" << std::endl;
    return 0;
  }

  if (argc == 2) read_parameters(argv[1]);

  integrate();

  return 0;
}

